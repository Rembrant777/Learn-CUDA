#include "hip/hip_runtime.h"
#include <cstdio>
#include <omp.h>
#include <helper_timer.h>

using namespace std; 

__global__ void vecAdd_kernel(float *c, const float *a, const float *b); 
void init_buffer(float *data, const int size); 

class Operator
{
private:
    int index; 
    hipStream_t stream; 
    StopWatchInterface *p_timer; 
    static void CUDART_CB Callback(hipStream_t stream, hipError_t status, void *userData);     
    void print_timer(); 
public:
    Operator() {
        hipStreamCreate(&stream); 
        sdkCreateTimer(&p_timer); 
    }    

    ~Operator() {
        hipStreamDestroy(stream); 
        sdkDeleteTimer(&p_timer);
    }

    void set_index(int idx) {
        index = idx; 
    }

    void async_operation(float *h_c, const float *h_a, const float *h_b,
                float *d_c, float *da, float *d_b,
                const int size, const int bufsize); 
}; // Operator 

void Operator::CUDART_CB Callback(hipStream_t stream, hipError_t status, void *userData)
{
    Operator* this_ = (Operator*) userData; 
    this_->print_timer(); 
}

void Operator::print_timer() 
{
    sdkStopTimer(&p_timer); 
    float elapsed_time_msed = sdkGetTimerValue(&p_timer); 
    printf("stream %2d - elapsed %.3f ms\n", index, elapsed_time_msed); 
}

void Operator::async_operation(float *h_c, const float *h_a, const float *h_b,
                    float *d_c, float *d_a, float *d_b,
                    const int size, const int bufsize)
{
    // start timer
    sdkStartTimer(&p_timer); 

    // copy host -> device 
    hipMemcpyAsync(d_a, h_a, bufsize, hipMemcpyHostToDevice, stream); 
    hipMemcpyAsync(d_b, h_b, bufsize, hipMemcpyHostToDevice, stream); 

    // launch cuda kernel 
    // 256 threads per block 
    dim3 dimBlock(256); 

    // size / 256 blocks per grid 
    dim3 dimGrid(size / dimBlock.x); 
    vecAdd_kernel<<<dimGrid, dimBlock, 0, stream>>>(d_c, d_a, d_b);

    // copy device -> host 
    hipMemcpyAsync(h_c, d_c, bufsize, hipMemcpyDeviceToHost, stream); 

    // register callback function 
    hipStreamAddCallback(stream, Operator::Callback, this, 0); 
}                    

int main(int argc, char *argv[])
{
    float *h_a, *h_b, *h_c; 
    float *d_a, *d_b, *d_c; 
    int size = 1 << 24; 
    int bufsize = size * sizeof(float); 
    int num_operator = 4; 

    if (argc != 1) {
        num_operator = atoi(argv[1]);
    }

    // initialize timer
    StopWatchInterface *timer; 
    sdkCreateTimer(&timer); 

    // allocate host memories 
    hipHostMalloc((void **)&h_a, bufsize); 
    hipHostMalloc((void **)&h_b, bufsize);
    hipHostMalloc((void **)&h_c, bufsize);

    // initialize host values 
    srand(2024); 
    init_buffer(h_a, size); 
    init_buffer(h_b, size); 
    init_buffer(h_c, size); 

    // allocate device memories 
    hipMalloc((void**) &d_a, bufsize); 
    hipMalloc((void**) &d_b, bufsize);
    hipMalloc((void**) &d_c, bufsize);

    // create list of operation elements
    Operator* ls_operator = new Operator[num_operator]; 

    sdkStartTimer(&timer); 

    // execute each operator correspoinding data
    // thread-resource allocated from host(cpu) side
    // block below will be executed in parallel
    // parallelism = max_threads_applied / total tasks 
    // this is cpu threads parallel unit
    omp_set_num_threads(num_operator); 
    #pragma omp parallel
    {
        // get current thread idx
        int i = omp_get_thread_num(); 
        // get init offset of to be computed array 
        int offset = i * size / num_operator; 

        // first set index of current thread to operator item 
        ls_operator[i].set_index(i);

        // here invoke cuda parallel operation 
        // this is cuda parallel unit 
        ls_operator[i].async_operation(&h_c[offset], &h_a[offset], &h_b[offset],
                            &d_c[offset], &d_a[offset], &d_b[offset],
                            size / num_operator, bufsize / num_operator); 
    }

    // sync all stream operation 
    hipDeviceSynchronize(); 
    sdkStopTimer(&timer);

    // print out the result 
    int print_idx = 256; 
    printf("compared a sample result ...\n");  
    printf("host: %.6f, device: %.6f\n", h_a[print_idx] + h_b[print_idx], h_c[print_idx]);

    // compute and print the performance 
    float elapsed_time_msed = sdkGetTimerValue(&timer); 
    float bandwidth = 3 * bufsize * sizeof(float) / elapsed_time_msed / 1e6; 
    printf("Time = %.3f msec, bandwidth = %f GB/s\n", elapsed_time_msed, bandwidth); 

    // delete timer
    sdkDeleteTimer(&timer); 

    // terminate operators 
    delete [] ls_operator; 

    // terminate device memories
    hipFree(d_a); 
    hipFree(d_b); 
    hipFree(d_c); 

    // terminate host memories 
    hipHostFree(h_a); 
    hipHostFree(h_b); 
    hipHostFree(h_c); 
    
    return 0; 
}

__global__ 
void vecAdd_kernel(float *c, const float* a, const float* b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0; i < 500; i++)
        c[idx] = a[idx] + b[idx];
}

void init_buffer(float *data, const int size)
{
    for (int i = 0; i < size; i++) 
        data[i] = rand() / (float)RAND_MAX;
}

