
#include <hip/hip_runtime.h>
#include <cstdio>

using namespace std; 

__global__ 
void foo_kernel(int step) 
{
    printf("loop: %d\n", step); 
}

int main() 
{
    int n_stream = 5; 
    hipStream_t *ls_stream; 
    ls_stream = (hipStream_t*) new hipStream_t[n_stream]; 

    // create multiple streams 
    for (int i = 0; i < n_stream; i++) {
        hipStreamCreate(&ls_stream[i]); 
    }

    // execute kernels with the CUDA streame each 
    for (int i = 0; i < n_stream; i++) {
        foo_kernel<<<1,1,0, ls_stream[i]>>>(i); 
        hipStreamSynchronize(ls_stream[i]); 
    }

    // synchornize the host and GPU 
    hipDeviceSynchronize(); 

    // terminates all the created CUDA streams 
    for (int i = 0; i < n_stream; i++) {
        hipStreamDestroy(ls_stream[i]); 
    }

    delete [] ls_stream; 

    return 0; 
}